#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <math.h>
#include <stdio.h>
#include "kernel.h"


__global__ void mulKernel(double* c, double* a, double* b,int n,int p) {
	int row = threadIdx.x+blockDim.x*blockIdx.x;
	if (row >= n) return;
	double temp = 0;
	for (int i = 0; i < p; i++) {
		temp += a[row * p + i] * b[i];
	}
	c[row] = temp;
}
__global__ void mulTransposeKernel(double* c, double* a, double* b,int n,int p,double alpha=1.0) {
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	if (col >= n) return;
	double temp = 0;
	for (int i = 0; i < n; i++) {
		temp += a[i * p + col] * b[i];
	}
	c[col] += temp*alpha;
}


__global__ void minusKernel(double* c, double* a, double* b,int n) {
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	if (index >= n) return;
	c[index] = a[index] - b[index];
}

__global__ void arrayXarray(double* c, double* a, double* b,int n) {
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	if (index >= n) return;
	c[index] = a[index] * b[index];
}

void tink4(double* theta,double* train_x,double* train_y,int n,int p,int blocksize) {
	double alpha = 0.0083;
	double *err_0,*y_pred,*dev_train_x,*dev_train_y,*dev_y_pred, *dev_theta,*dev_err,*dev_err2;
	for (int i = 0; i < p; i++)
		theta[i] = (rand() % 10) / 1000.0;

	hipMalloc((void**)& dev_theta,p * sizeof(double));
	hipMalloc((void**)& dev_train_x, n*p * sizeof(double));
	hipMalloc((void**)& dev_train_y, n* sizeof(double));
	hipMalloc((void**)& dev_y_pred, n* sizeof(double));
	hipMalloc((void**)& dev_err, n* sizeof(double));
	hipMalloc((void**)& dev_err2, n* sizeof(double));

	hipMemcpy(dev_train_x, train_x, p * n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_train_y, train_y, n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_theta, theta, p * sizeof(double), hipMemcpyHostToDevice);

	err_0 = (double*)malloc(n * sizeof(double));
	
	int cnt = 2000000, it = 0;
	double last_error, error = 100;
	do {
		last_error = error;
		double error2 = 0;

		mulKernel <<< ceil(n/blocksize),blocksize >> > (dev_y_pred, dev_train_x, dev_theta, n,p);

		hipDeviceSynchronize();
		minusKernel << <ceil(n/blocksize),blocksize>> > (dev_err, dev_y_pred, dev_train_y,n);
		hipDeviceSynchronize();
		arrayXarray << <ceil(n/blocksize),blocksize>> > (dev_err2, dev_err, dev_err,n);

		hipDeviceSynchronize();
		mulTransposeKernel << <1,p >> > (dev_theta, dev_train_x, dev_err,n,p,-alpha/n);

		hipDeviceSynchronize();
		hipMemcpy(err_0, dev_err2, n * sizeof(double), hipMemcpyDeviceToHost);
		for (int i = 0; i < n; i++) error2 += err_0[i];

		error = sqrt(error2);
		if (error > last_error)
			alpha /= 1.2;
		hipDeviceSynchronize();

	} while (it++ < cnt && fabs(error - last_error) > epsilon);
	printf("in %d iterations \n", it);
	hipMemcpy(theta, dev_theta, p * sizeof(double), hipMemcpyDeviceToHost);
	hipFree(dev_err);
	hipFree(dev_train_x);
	hipFree(dev_train_y);
	hipFree(dev_y_pred);
	hipFree(dev_theta);
}
